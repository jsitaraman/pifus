#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <hipblas.h>

#define HASH_64

#ifdef HASH_64
#define UINT_TYPE uint64_t
#define REAL double
#else
#define UINT_TYPE uint32_t
#define REAL float
#endif	


__global__ void arange(int* a, int n){
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i<n) a[i] = i;
}

#ifdef HASH_64
__inline__ __device__ UINT_TYPE expand_bits(UINT_TYPE v){
  v = (v * 0x000100000001u) & 0xFFFF00000000FFFFu;
  v = (v * 0x000000010001u) & 0x00FF0000FF0000FFu;
  v = (v * 0x000000000101u) & 0xF00F00F00F00F00Fu;
  v = (v * 0x000000000011u) & 0x30C30C30C30C30C3u;
  v = (v * 0x000000000005u) & 0x9249249249249249u;
  return v;
}
#else
__inline__ __device__ UINT_TYPE expand_bits(UINT_TYPE v){
  v = (v * 0x00010001u) & 0xFF0000FFu;
  v = (v * 0x00000101u) & 0x0F00F00Fu;
  v = (v * 0x00000011u) & 0xC30C30C3u;
  v = (v * 0x00000005u) & 0x49249249u;
  return v;
}
#endif

__global__ void compute_morton(int total, UINT_TYPE *mkeys, REAL (*xyz)[3], REAL* box){

  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < total){
	
    REAL x, y, z;
    x = (xyz[idx][0]-box[0])/(box[3]-box[0]);
    y = (xyz[idx][1]-box[1])/(box[4]-box[1]);
    z = (xyz[idx][2]-box[2])/(box[5]-box[2]);
		
    UINT_TYPE mx, my, mz;
#ifdef HASH_64
    x *= 1024*1024-1;
    y *= 1024*1024-1;
    z *= 1024*1024-1;
#else
    x *= 1023.0f;
    y *= 1023.0f;
    z *= 1023.0f;
#endif
    mx = expand_bits((UINT_TYPE)x);
    my = expand_bits((UINT_TYPE)y);
    mz = expand_bits((UINT_TYPE)z);
		
    mkeys[idx] = mx * 4 + my * 2 + mz;
  }
}

void nodes_compute_morton(int total, UINT_TYPE *mkeys, REAL (*xyz)[3], REAL* box){
  dim3 threads(256,1,1);
  dim3 blocks(1,1,1);
  blocks.x = (total-1)/threads.x+1;
  compute_morton<<<blocks,threads>>>(total, mkeys, xyz, box);
}

__global__ void setbox(REAL* xyz, REAL* box, int* ibox){
  int i=threadIdx.x;
  if(i<6) box[i] = xyz[ibox[i]*3+i];
}

__global__ void dbgprint(REAL* x, int n){
  int i;
  for(i=0; i<n; i++){
    printf("__GPU__ %16.8e %16.8e %16.8e\n", x[i*3+0],x[i*3+1],x[i*3+2]);
  }
}

__global__ void dbgiprint(int* x, int n){
  int i;
  for(i=0; i<n; i++){
    printf("__GPU__ %d\n", x[i]);
  }
}

__global__ void dbguprint(UINT_TYPE* x, int n){
  int i;
  for(i=0; i<n; i++){
    printf("__GPU__ %6d : %lu\n", i, x[i]);
  }
}

void getbox(REAL* xyz, int n, REAL* box){
  
  // Create a handle for CUBLAS
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  int *c_ibox = new int[6];
  int *d_ibox;
  hipMalloc((void**)&d_ibox, 6*sizeof(int));

#ifdef HASH_64
  hipblasIdamin(handle, n, &xyz[0], 1, &c_ibox[0]);
  hipblasIdamin(handle, n, &xyz[1], 1, &c_ibox[1]);
  hipblasIdamin(handle, n, &xyz[2], 1, &c_ibox[2]);
  hipblasIdamax(handle, n, &xyz[0], 1, &c_ibox[3]);
  hipblasIdamax(handle, n, &xyz[1], 1, &c_ibox[4]);
  hipblasIdamax(handle, n, &xyz[2], 1, &c_ibox[5]);
#else
  hipblasIsamin(handle, n, &xyz[0], 3, &c_ibox[0]);
  hipblasIsamin(handle, n, &xyz[1], 3, &c_ibox[1]);
  hipblasIsamin(handle, n, &xyz[2], 3, &c_ibox[2]);
  hipblasIsamax(handle, n, &xyz[0], 3, &c_ibox[3]);
  hipblasIsamax(handle, n, &xyz[1], 3, &c_ibox[4]);
  hipblasIsamax(handle, n, &xyz[2], 3, &c_ibox[5]);
#endif

  hipMemcpy(d_ibox, c_ibox, 6*sizeof(int), hipMemcpyHostToDevice);

  setbox<<<1,6>>>(xyz,box,d_ibox);
  
  // Destroy the handle
  hipblasDestroy(handle);

  hipFree(d_ibox);
  delete[] c_ibox;
  
}
  
void msort(REAL* xyz, int& n, int* isorted){

  UINT_TYPE*  d_m;
  REAL* box;
  hipMalloc((void**)&box, 6*sizeof(REAL));
  hipMalloc((void**)&d_m, n*sizeof(UINT_TYPE));
  
  getbox(xyz,n,box);

  // git morton indices
  nodes_compute_morton(n, d_m, (REAL (*)[3])xyz, box);

  // UINT_TYPE* c_m = new UINT_TYPE[n];
  // hipMemcpy(c_m, d_m, n*sizeof(UINT_TYPE), hipMemcpyDeviceToHost);
  // for(int i=0; i<n; i++){
  //   printf("%6d : %lu\n", i, c_m[i]);
  // }
  // dbguprint<<<1,1>>>(d_m, n);
  // hipDeviceSynchronize();
  
  // set the array indices to simple increment
  arange<<<(n-1)/256+1,256>>>(isorted,n);

  thrust::device_ptr<UINT_TYPE> thm( d_m );
  thrust::device_ptr<int>  thi( isorted );
  thrust::sort_by_key( thm, thm+n, thi ); // sort by the index

  hipFree(box);
  hipFree(d_m);

}











