#include <hip/hip_runtime.h>
#include "dMeshBlock.h"
#include "cuda_macros.h"

void dMeshBlock::setData(int btag_in,int nnodes_in,double *xin)
{
    btag=btag_in;
    nnodes=nnodes_in;
    int xsize=3*nnodes*sizeof(double);
    pushToDevice(x,xin,xsize);
    /*
    hipMalloc((void **) &x,xsize);
    hipMemcpy(x,xin,xsize,hipMemcpyHostToDevice);
    */
}


void dMeshBlock::setQ(int nvar,double *qin) {
    int qsize=nvar*nnodes*sizeof(double);
    pushToDevice(q,qin,qsize);
    /*
    hipMalloc((void **) &q,qsize);
    hipMemcpy(q,qin,qsize,hipMemcpyHostToDevice);
    */
}

void dMeshBlock::setTargets(int nvar,int ntargets_in,double *xtargets_in,double *qtargets_in) 
{
  ntargets=ntargets_in;
  int xsize=3*ntargets*sizeof(double);
  pushToDevice(xtarget,xtargets_in,xsize);
  /*
  hipMalloc((void **) &xtarget,xsize);
  hipMemcpy(xtarget,xtargets_in,xsize,hipMemcpyHostToDevice);
  */
  int qsize=nvar*nnodes*sizeof(double);
  pushToDevice(qtarget,qtargets_in,qsize);
  /*
  hipMalloc((void **) &qtarget,qsize);
  hipMemcpy(qtarget,qtargets_in,qsize,hipMemcpyHostToDevice);
  */
}

dMeshBlock::~dMeshBlock() {
   if (elementBbox) hipFree(elementBbox);
   if (pcount) hipFree(pcount);
   if (pindx) {
     for(int i=0;i<ntargets;i++) hipFree(pindx[i]);
     for(int i=0;i<ntargets;i++) hipFree(weights[i]);
     hipFree(pindx);
     hipFree(weights);
   }
   delete [] dadt;
}

void dMeshBlock::preprocess(ADT *adt)
{
  if (dadt)
   {
     dadt->clearData();
   }
  else		
   {
     dadt=new dADT[1];
   }

  int *adtIntegers=adt->getadtIntegers();
  int nelem=adt->getadtnelem();
  double *adtReals=adt->getadtReals();
  double *adtExtents=adt->getadtExtents();
  double *coord=adt->getadtcoord();

  dadt->setHostTree(nelem,adtIntegers,adtReals,adtExtents,coord);
}
