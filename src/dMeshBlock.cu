#include "hip/hip_runtime.h"
#include "dMeshBlock.h"
#include "dADT.h"
#include "ADT.h"

#include "pifus_types.h"
#include "pifus_cuda.h"
#include "hip/device_functions.h"

// __device__ __constant__  static double v0[3][3] = {1.0,0.0,0.0,
//                                                    0.0,1.0,0.0,
//                                                    0.0,0.0,1.0};

// __constant__ double v0[3][3];

__global__
void d_searchADTRegion(int ndim,int nelem,
                       double *x, double *xtarget, int* isorted, 
                       int *adtIntegers, double *adtReals,
		       double *adtExtents, 
                       double *coord,int *ndes,int *pcount, 
		       int *pindx, double *weights, int ntargets)
{
 int i0 = blockIdx.x * blockDim.x + threadIdx.x;

//TRACEI(ntargets);
//for(int idx=0;idx < ntargets;idx++)
 if (i0 < ntargets)
 {
   int idx = i0; //isorted[i0];
  //printf("---- GPU search --- \n");
  double vec[9];
  double xcloud[24];
  int indx[9];	// 8+1 ... the +1 is a quick fix for memory issues
  int np=8;
  int nchecks;
  int p=0;
  double dmin[2];
  double v0[3][3] = {1.0,0.0,0.0,
                     0.0,1.0,0.0,
                     0.0,0.0,1.0};
  for(int j=0;j<24;j++) xcloud[j]=-1000;
  //
  /*
  printf("%f %f %f\n",xtarget[3*idx],xtarget[3*idx+1],xtarget[3*idx+2]);
  for(int j=0;j < 10; j++)
   printf("%d \n",adtIntegers[j]);
  */
  //
  for(int l=1;l>=-1;l-=2)
     for(int k=1;k>=-1;k-=2)
        for(int j=1;j>=-1;j-=2)
            {
              int m=0;
              for(int n=0;n<3;n++)
                {
                  vec[m++]=v0[n][0]*j;
                  vec[m++]=v0[n][1]*k;
                  vec[m++]=v0[n][2]*l;
                }
                nchecks=0;
                indx[p]=indx[p+1]=-1;
		dmin[0]=dmin[1]=PIFUS_BIGVALUE;
		//searchIntersections_region_norecursion(&(indx[p]),adtIntegers,adtReals,
		//	     coord,0,0,dmin,&(xtarget[3*idx]),vec,nelem,ndim,
                //             &nchecks);
		//d_searchIntersections_norecursion(&(indx[p]),adtIntegers,adtReals,coord,
                //                            &(xtarget[3*idx]),vec,nelem,ndim,&nchecks);

                d_searchIntersections_norecursion_nostack(&(indx[p]),adtIntegers,adtReals,coord,ndes,
                                                 &(xtarget[3*idx]),vec,nelem,ndim,&nchecks);

		printf("pointIndex, nchecks=%d %d\n",indx[p],nchecks);
              p++;
            }
    int m=0;
    TRACED(xtarget[3*idx]);
    TRACED(xtarget[3*idx+1]);
    TRACED(xtarget[3*idx+2]);
    for(p=0;p<np;p++)
     {
       if (indx[p] > -1) {
         xcloud[m++]=x[3*indx[p]];
         xcloud[m++]=x[3*indx[p]+1];
	 xcloud[m++]=x[3*indx[p]+2];
         }
     }
    int itype=1;
    int iflag;
    pcount[idx]=m/3;
    //TRACEI(pcount[idx]);
    for(p=0;p<m/3;p++) {
     printf("xcloud: %f %f %f\n",xcloud[3*p],xcloud[3*p+1],xcloud[3*p+2]);
    }
    //printf("pointer=%p\n",xcloud);
    d_interprbf(xcloud,&(xtarget[3*idx]),&(weights[8*idx]),pcount[idx],itype,&iflag);
    for(p=0;p<m/3;p++) pindx[8*idx+p]=indx[p];
    //for(p=0;p<pcount[idx];p++) TRACED(weights[8*idx+p]);
 }
}

__global__
void d_interpolate(int nvar,int ntargets,double *q,double *qtarget, int *pcount, int *pindx, double *weights)
{
 int idx = blockIdx.x * blockDim.x + threadIdx.x;
 if (idx < ntargets)	
 // for(int idx=0;idx<ntargets;idx++)
    {
      for(int j=0;j<nvar;j++)
	{
	  qtarget[idx*nvar+j]=0.0;
	  for(int p=0;p<pcount[idx];p++)
            {
            //printf("%d %f %f\n",pindx[8*idx+p],weights[8*idx+p],q[pindx[8*idx+p]*nvar+j]);
	    qtarget[idx*nvar+j]+=(weights[8*idx+p]*q[pindx[8*idx+p]*nvar+j]);
            }
	}
    }
}

namespace PIFUS {

void dMeshBlock::setData(int btag_in,int nnodes_in,double *xin)
{
    btag=btag_in;
    nnodes=nnodes_in;
    int xsize=3*nnodes*sizeof(double);
    pushToDeviceDouble(x,xin,xsize);
    /*
    hipMalloc((void **) &x,xsize);
    hipMemcpy(x,xin,xsize,hipMemcpyHostToDevice);
    */
}


void dMeshBlock::setQ(int nvar,double *qin) {
    int qsize=nvar*nnodes*sizeof(double);
    pushToDeviceDouble(q,qin,qsize);
    /*
    hipMalloc((void **) &q,qsize);
    hipMemcpy(q,qin,qsize,hipMemcpyHostToDevice);
    */
}

void dMeshBlock::setTargets(int nvar,int ntargets_in,double *xtargets_in,double *qtargets_in) 
{
  ntargets=ntargets_in;
  //TRACEI(ntargets);
  int xsize=3*ntargets*sizeof(double);
  pushToDeviceDouble(xtarget,xtargets_in,xsize);
  /*
  hipMalloc((void **) &xtarget,xsize);
  hipMemcpy(xtarget,xtargets_in,xsize,hipMemcpyHostToDevice);
  */
  int qsize=nvar*nnodes*sizeof(double);
  pushToDeviceDouble(qtarget,qtargets_in,qsize);
  qH=qtargets_in;
  /*
  hipMalloc((void **) &qtarget,qsize);
  hipMemcpy(qtarget,qtargets_in,qsize,hipMemcpyHostToDevice);
  */
}

void dMeshBlock::search(void)
{
 if (weights!=NULL || pindx !=NULL) {
   deallocateDevice(weights);
   deallocateDevice(pindx);
   deallocateDevice(pcount);
 }
 
 allocateOnDeviceDouble(weights,8*ntargets*sizeof(double));
 allocateOnDeviceInt(pindx,8*ntargets*sizeof(int));
 allocateOnDeviceInt(pcount,ntargets*sizeof(int));

 int block_size = 128;
 int n_blocks = ntargets/block_size + (ntargets%block_size == 0 ? 0:1);

 // double v0_cpu[3][3] = {1.0,0.0,0.0,
 //                        0.0,1.0,0.0,
 //                        0.0,0.0,1.0};
 // hipMemcpyToSymbol(HIP_SYMBOL(v0), v0_cpu, 9*sizeof(double));

 int* isorted = NULL;
 //allocateOnDeviceInt(isorted,ntargets*sizeof(int)); 
 //msort(xtarget, ntargets, isorted);

 d_searchADTRegion<<< n_blocks, block_size >>> 
                  (dadt->ndim,
                   dadt->nelem,
                   x,xtarget,
                   isorted,
                   dadt->adtIntegers,
		   dadt->adtReals,
                   dadt->adtExtents,
                   dadt->coord,
                   dadt->ndescendents,
                   pcount,
                   pindx,
                   weights,
                   ntargets);
  hipDeviceSynchronize();

  //deallocateDevice(isorted);


 /*
 d_searchADTRegion(dadt->ndim,
                   dadt->nelem,
                   x,xtarget,
                   dadt->adtIntegers,
		   dadt->adtReals,
                   dadt->adtExtents,
                   dadt->coord,
                   pcount,
                   pindx,
                   weights,
                   ntargets);
 */

}
void dMeshBlock::interpolate(int nvar)
{
 int block_size = 512;
 int n_blocks = ntargets/block_size + (ntargets%block_size == 0 ? 0:1);
 d_interpolate <<< n_blocks, block_size >>> (nvar,ntargets,q,qtarget,pcount,pindx,weights);
 //d_interpolate(nvar,ntargets,q,qtarget,pcount,pindx,weights);
 //for(int i=0;i<nvar*ntargets;i++) printf("%f ",qH[i]);
 //printf("\n");
 pullFromDevice(qH,qtarget,sizeof(double)*nvar*ntargets);
 //for(int i=0;i<nvar*ntargets;i++) printf("%f ",qH[i]);
 //printf("\n");
}

void dMeshBlock::preprocess(ADT *adt)
{
  if (dadt)
   {
     dadt->clearData();
   }
  else		
   {
     dadt=new dADT[1];
   }

  int *adtIntegers=adt->getadtIntegers();
  int nelem=adt->getadtnelem();
  double *adtReals=adt->getadtReals();
  double *adtExtents=adt->getadtExtents();
  double *coord=adt->getadtcoord();
  int *ndesc=adt->getdescendents();

  dadt->setHostTree(nelem,adtIntegers,adtReals,adtExtents,coord,ndesc);
}


dMeshBlock::~dMeshBlock() {
   if (elementBbox) deallocateDevice(elementBbox);
   if (pcount) deallocateDevice(pcount);
   if (pindx) {
     deallocateDevice(pindx);
     deallocateDevice(weights);
   }
   delete [] dadt;
}

} // namespace PIFUS
