#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "dMeshBlock.h"
#include "cuda_macros.h"
#include "hip/device_functions.h"


//extern "C" {
//void searchIntersections_region_norecursion(int *pointIndex,int *adtIntegers,double *adtReals,
//				double *coord,int level,int node,double *dmin,
//				double *xsearch,double *vec,int nelem,int ndim,int *nchecks);
//}


__global__ 
void d_searchADTRegion(int ndim,int nelem,
     double *x, double *xtarget, int *adtIntegers, double *adtReals,double *adtExtents, 
     double *coord,int *pcount, int *pindx, double *weights, int ntargets)
{
 int idx = blockIdx.x * blockDim.x + threadIdx.x;
//TRACEI(ntargets);
//for(int idx=0;idx < ntargets;idx++)
 if (idx < ntargets)
 {
  //printf("---- GPU search --- \n");
  double v0[3][3];
  double vec[9];
  double xcloud[24];
  int indx[8];	
  int np=8;
  int nchecks;
  int p=0;
  double dmin[2];
  //
  v0[0][0]=1.0;v0[1][0]=0.0;;v0[2][0]=0.0;
  v0[0][1]=0.0;v0[1][1]=1.0;;v0[2][1]=0.0;
  v0[0][2]=0.0;v0[1][2]=0.0;;v0[2][2]=1.0;
  //
  /*
  printf("%f %f %f\n",xtarget[3*idx],xtarget[3*idx+1],xtarget[3*idx+2]);
  for(int j=0;j < 10; j++)
   printf("%d \n",adtIntegers[j]);
  */
  //
  for(int l=1;l>=-1;l-=2)
     for(int k=1;k>=-1;k-=2)
        for(int j=1;j>=-1;j-=2)
            {
              int m=0;
              for(int n=0;n<3;n++)
                {
                  vec[m++]=v0[n][0]*j;
                  vec[m++]=v0[n][1]*k;
                  vec[m++]=v0[n][2]*l;
                }
                nchecks=0;
                indx[p]=indx[p+1]=-1;
		dmin[0]=dmin[1]=PIFUS_BIGVALUE;
		//searchIntersections_region_norecursion(&(indx[p]),adtIntegers,adtReals,
		//	     coord,0,0,dmin,&(xtarget[3*idx]),vec,nelem,ndim,
                //             &nchecks);
		d_searchIntersections_norecursion(&(indx[p]),adtIntegers,adtReals,coord,
                                                 &(xtarget[3*idx]),vec,nelem,ndim,&nchecks);
		//printf("pointIndex, nchecks=%d %d\n",indx[p],nchecks);
              p++;
            }
    int m=0;
    for(p=0;p<np;p++)
     {
       //TRACEI(indx[p]);
       if (indx[p] > -1) {
         xcloud[m++]=x[3*indx[p]];
         xcloud[m++]=x[3*indx[p]+1];
	 xcloud[m++]=x[3*indx[p]+2];
         }
     }
    int itype=1;
    int iflag;
    pcount[idx]=m/3;
    for(p=0;p<m/3;p++) pindx[8*idx+p]=indx[p];
    d_interprbf(xcloud,&(xtarget[3*idx]),&(weights[8*idx]),pcount[idx],itype,&iflag);
    //for(p=0;p<pcount[idx];p++) TRACED(weights[8*idx+p]);
 }
}

__global__
void d_interpolate(int nvar,int ntargets,double *q,double *qtarget, int *pcount, int *pindx, double *weights)
{
 int idx = blockIdx.x * blockDim.x + threadIdx.x;
 if (idx < ntargets)	
 // for(int idx=0;idx<ntargets;idx++)
    {
      for(int j=0;j<nvar;j++)
	{
	  qtarget[idx*nvar+j]=0.0;
	  for(int p=0;p<pcount[idx];p++)
            {
            //printf("%d %f %f\n",pindx[8*idx+p],weights[8*idx+p],q[pindx[8*idx+p]*nvar+j]);
	    qtarget[idx*nvar+j]+=(weights[8*idx+p]*q[pindx[8*idx+p]*nvar+j]);
            }
	}
    }
}

void dMeshBlock::setData(int btag_in,int nnodes_in,double *xin)
{
    btag=btag_in;
    nnodes=nnodes_in;
    int xsize=3*nnodes*sizeof(double);
    pushToDeviceDouble(x,xin,xsize);
    /*
    hipMalloc((void **) &x,xsize);
    hipMemcpy(x,xin,xsize,hipMemcpyHostToDevice);
    */
}


void dMeshBlock::setQ(int nvar,double *qin) {
    int qsize=nvar*nnodes*sizeof(double);
    pushToDeviceDouble(q,qin,qsize);
    /*
    hipMalloc((void **) &q,qsize);
    hipMemcpy(q,qin,qsize,hipMemcpyHostToDevice);
    */
}

void dMeshBlock::setTargets(int nvar,int ntargets_in,double *xtargets_in,double *qtargets_in) 
{
  ntargets=ntargets_in;
  //TRACEI(ntargets);
  int xsize=3*ntargets*sizeof(double);
  pushToDeviceDouble(xtarget,xtargets_in,xsize);
  /*
  hipMalloc((void **) &xtarget,xsize);
  hipMemcpy(xtarget,xtargets_in,xsize,hipMemcpyHostToDevice);
  */
  int qsize=nvar*nnodes*sizeof(double);
  pushToDeviceDouble(qtarget,qtargets_in,qsize);
  qH=qtargets_in;
  /*
  hipMalloc((void **) &qtarget,qsize);
  hipMemcpy(qtarget,qtargets_in,qsize,hipMemcpyHostToDevice);
  */
}

void dMeshBlock::search(void)
{
 if (weights!=NULL || pindx !=NULL) {
   deallocateDevice(weights);
   deallocateDevice(pindx);
   deallocateDevice(pcount);
 }
 
 allocateOnDeviceDouble(weights,8*ntargets*sizeof(double));
 allocateOnDeviceInt(pindx,8*ntargets*sizeof(int));
 allocateOnDeviceInt(pcount,ntargets*sizeof(int));

 int block_size = 512;
 int n_blocks = ntargets/block_size + (ntargets%block_size == 0 ? 0:1);
 d_searchADTRegion<<< n_blocks, block_size >>> 
                  (dadt->ndim,
                   dadt->nelem,
                   x,xtarget,
                   dadt->adtIntegers,
		   dadt->adtReals,
                   dadt->adtExtents,
                   dadt->coord,
                   pcount,
                   pindx,
                   weights,
                   ntargets);
  hipDeviceSynchronize();


 /*
 d_searchADTRegion(dadt->ndim,
                   dadt->nelem,
                   x,xtarget,
                   dadt->adtIntegers,
		   dadt->adtReals,
                   dadt->adtExtents,
                   dadt->coord,
                   pcount,
                   pindx,
                   weights,
                   ntargets);
 */

}
void dMeshBlock::interpolate(int nvar)
{
 int block_size = 512;
 int n_blocks = ntargets/block_size + (ntargets%block_size == 0 ? 0:1);
 d_interpolate <<< n_blocks, block_size >>> (nvar,ntargets,q,qtarget,pcount,pindx,weights);
 //d_interpolate(nvar,ntargets,q,qtarget,pcount,pindx,weights);
 //for(int i=0;i<nvar*ntargets;i++) printf("%f ",qH[i]);
 //printf("\n");
 pullFromDevice(qH,qtarget,sizeof(double)*nvar*ntargets);
 //for(int i=0;i<nvar*ntargets;i++) printf("%f ",qH[i]);
 //printf("\n");
}

void dMeshBlock::preprocess(ADT *adt)
{
  if (dadt)
   {
     dadt->clearData();
   }
  else		
   {
     dadt=new dADT[1];
   }

  int *adtIntegers=adt->getadtIntegers();
  int nelem=adt->getadtnelem();
  double *adtReals=adt->getadtReals();
  double *adtExtents=adt->getadtExtents();
  double *coord=adt->getadtcoord();

  dadt->setHostTree(nelem,adtIntegers,adtReals,adtExtents,coord);
}


dMeshBlock::~dMeshBlock() {
   if (elementBbox) deallocateDevice(elementBbox);
   if (pcount) deallocateDevice(pcount);
   if (pindx) {
     deallocateDevice(pindx);
     deallocateDevice(weights);
   }
   delete [] dadt;
}
