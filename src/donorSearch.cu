// -*- c++ -*- 
//#include "precision.h"
#define BIGVAL 1E6	
#define TOL 1e-7

#include <hip/hip_runtime.h>
#include<math.h>
#include<stdio.h>
#define REAL double
__device__ int faceEdgeIntersectCheck(REAL xface[4][3],
			   REAL xx[3],
			   REAL xp[3],
			   REAL xi[3],
			   REAL EPS1,
			   REAL EPS2,
			   int nvert);
__global__ void donorSearch(int *ndc4,
		 int *ndc5,
		 int *ndc6,
		 int *ndc8,
		 int *neig,
		 REAL *x,
		 int *auxGrid,
		 int mdimx,int mdimy,int mdimz,
		 REAL xminx,REAL xminy, REAL xminz,
		 REAL dsx,REAL dsy, REAL dsz,	    
		 REAL *cellCenter,
		 REAL *scfac,
		 int ntetra,
		 int npyra,
		 int nprizm,
		 int nhexa,
		 int ncells,
		 int *interp,
		 int *fringe,
		 REAL *deltax,
		 REAL *xsearch,
		 REAL *rcap,
		 int *iblank,
		 int nsearch)
                 //!int* compactionIndex,
                 //!int* recipient)
{
  int idx=blockIdx.x*blockDim.x + threadIdx.x;
  int e[8],i,j,k,nvert;
  int icell,nfaces,ipoly;
  REAL xx[3],xp[3],xi[3];
  int numverts[4][6]={3,3,3,3,0,0,4,3,3,3,3,0,3,4,4,4,3,0,4,4,4,4,4,4};
  int faceInfo[4][24]={1,2,3,0,1,4,2,0,2,4,3,0,1,3,4,0,0,0,0,0,0,0,0,0,
		       1,2,3,4,1,5,2,0,2,5,3,0,4,3,5,0,1,4,5,0,0,0,0,0,
		       1,2,3,0,1,4,5,2,2,5,6,3,1,3,6,4,4,6,5,0,0,0,0,0,
		       1,2,3,4,1,5,6,2,2,6,7,3,3,7,8,4,1,4,8,5,5,8,7,6};
  int indx,findx;
  int idonor;
  int searchComplete;
  int donorPrev;
  int inside,intersect;
  REAL xface[4][3];
  int mdim[3];
  REAL ds[3];
  REAL xmin[3];
  // FILE *fp;

  if (idx < nsearch)
    {
      xmin[0]=xminx;
      xmin[1]=xminy;
      xmin[2]=xminz;
      ds[0]=dsx;
      ds[1]=dsy;
      ds[2]=dsz;
      mdim[0]=mdimx;
      mdim[1]=mdimy;
      mdim[2]=mdimz;
      //
      inside=1;
      //
    xx[0]=xsearch[3*idx];
    xx[1]=xsearch[3*idx+1];
    xx[2]=xsearch[3*idx+2];
    //
    k=(int)((xx[2]-xmin[2])/ds[2]);
    j=(int)((xx[1]-xmin[1])/ds[1]);
    i=(int)((xx[0]-xmin[0])/ds[0]);
    //
    if ( i < 0 || i > mdim[0]-1) inside=0;
    if ( j < 0 || j > mdim[1]-1) inside=0;
    if ( k < 0 || k > mdim[2]-1) inside=0;
    //
  
    if (inside) {
      indx=k*mdim[1]*mdim[0]+j*mdim[0]+i;
      idonor=auxGrid[indx];
      searchComplete=0;
      donorPrev=-3;
      if (idonor >=0) {
	xp[0]=cellCenter[3*idonor];
	xp[1]=cellCenter[3*idonor+1];
	xp[2]=cellCenter[3*idonor+2];
	
	while(idonor >=0 && !searchComplete) 
	  {
	    if (idonor < ntetra) {
	      icell=4*idonor;
	      nvert=4;
	      nfaces=4;
	      ipoly=0;
	      for (k=0;k<nvert;k++)
		e[k]=ndc4[icell+k];
	    }
	    else if (idonor < ntetra+npyra) {
	      nvert=5;
	      nfaces=5;
	      icell=nvert*(idonor-ntetra);
	      ipoly=1;
	      for (k=0;k<nvert;k++)
		e[k]=ndc5[icell+k];
	    }
	    else if (idonor < ntetra+npyra+nprizm) {
	      nvert=6;
	      nfaces=5;
	      ipoly=2;
	      icell=nvert*(idonor-ntetra-npyra);
	      for (k=0;k< nvert;k++)
		e[k]=ndc6[icell+k];
	    }
	    else if (idonor < ntetra+npyra+nprizm+nhexa){
	      nvert=8;
	      nfaces=6;
	      ipoly=3;
	      icell=nvert*(idonor-ntetra-npyra-nprizm);
	      for(k=0;k<nvert;k++)
		e[k]=ndc8[icell+k];
	    }
	    
	    k=0;
	    intersect=0;
	    
// 	    fp=fopen("cell.dat","w");
// 	    fprintf(fp,"TITLE=cell file\n");
// 	    fprintf(fp,"VARIABLES=\"X\",\"Y\",\"Z\"\n");
// 	    fprintf(fp,"ZONE T=\"VOL_MIXED\",N=6 E=1 ET=BRICK, F=FEPOINT\n");
// 	    for(j=0;j<nvert;j++)
// 	      fprintf(fp,"%f %f %f\n",x[3*e[j]],x[3*e[j]+1],x[3*e[j]+2]);
// 	    fprintf(fp,"1 2 3 3 4 5 6 6\n");
// 	    fprintf(fp,"ZONE\n");
// 	    fprintf(fp,"%f %f %f\n",xx[0],xx[1],xx[2]);
// 	    fprintf(fp,"%f %f %f\n",xp[0],xp[1],xp[2]);
// 	    fclose(fp);

	    k=0;
	    while(k<nfaces && !intersect)
	      {
		if (neig[6*idonor+k]!=donorPrev && !intersect)
		  {
		    for(j=0;j<numverts[ipoly][k];j++)
		      {
			findx=e[faceInfo[ipoly][4*k+j]-1];
			xface[j][0]=x[3*findx];
			xface[j][1]=x[3*findx+1];
			xface[j][2]=x[3*findx+2];
		      }		
		    intersect=faceEdgeIntersectCheck(xface,xx,xp,xi,-TOL,TOL,
						     numverts[ipoly][k]);
		    if (intersect) {
		      donorPrev=idonor;
		      idonor=neig[6*idonor+k];
		      xp[0]=xi[0];
		      xp[1]=xi[1];
		      xp[2]=xi[2];
		    }
		  }
		k++;
	      }
	    
	    if (!intersect){
	      if (scfac[idonor] < rcap[idx]) {
		interp[idonor]=idx;
                fringe[0+2*idx]=idonor;
                fringe[1+2*idx]=idx;
                //! fringe[idx] = idonor

                //!compactionIndex[idx]=0;
                //!recipient[idx] = idx;
		//deltax[3*idonor]=xx[0]-cellCenter[3*idonor];
		//deltax[3*idonor+1]=xx[1]-cellCenter[3*idonor+1];
		//deltax[3*idonor+2]=xx[2]-cellCenter[3*idonor+2];
		deltax[3*idx]=xx[0]-cellCenter[3*idonor];
		deltax[3*idx+1]=xx[1]-cellCenter[3*idonor+1];
		deltax[3*idx+2]=xx[2]-cellCenter[3*idonor+2];
		iblank[idx]=-1;
	      }
	      else {
		iblank[idx]=-2;
	      }	    
	      searchComplete=1;
	  }
	}
      }
    }
  }    
}	

__device__ int faceEdgeIntersectCheck(REAL xface[4][3],
			   REAL xx[3],
			   REAL xp[3],
			   REAL xi[3],
			   REAL EPS1,
			   REAL EPS2,
			   int nvert)
{
  REAL mat[3][3];
  REAL rhs[3];
  REAL sol[3];
  REAL ONE1=1+EPS1;
  REAL ONE2=1+EPS2;
  int i,j;
  REAL det,deti;
//  FILE *fp;

//   fp=fopen("cell.dat","a");
//   fprintf(fp,"ZONE T=\"VOL_MIXED\",N=%d E=1 ET=QUADRILATERAL, F=FEPOINT\n",nvert);  
//   for(i=0;i<nvert;i++)
//     fprintf(fp,"%f %f %f\n",xface[i][0],xface[i][1],xface[i][2]);
//   fprintf(fp,"1 2 3 %d\n",(nvert==4)?nvert:3);
//   fclose(fp);

  for(i=1;i<nvert-1;i++)
    {
      
      for (j=0;j<3;j++)
	{
	  mat[j][0]=xp[j]-xx[j];
	  mat[j][1]=xface[i][j]-xface[0][j];
	  mat[j][2]=xface[i+1][j]-xface[0][j];
	  rhs[j]=xp[j]-xface[0][j];
	}
      //
      // invert
      // 
      det=( mat[0][0]*(mat[2][2]*mat[1][1]-mat[2][1]*mat[1][2])-
	    mat[1][0]*(mat[2][2]*mat[0][1]-mat[2][1]*mat[0][2])+
	    mat[2][0]*(mat[1][2]*mat[0][1]-mat[1][1]*mat[0][2]));

      if (fabs(det) > 1E-10) {
	deti=1./det;
	sol[0]=deti*
	  ( (mat[2][2]*mat[1][1]-mat[2][1]*mat[1][2])*rhs[0]
	    -(mat[2][2]*mat[0][1]-mat[2][1]*mat[0][2])*rhs[1]
	    +(mat[1][2]*mat[0][1]-mat[1][1]*mat[0][2])*rhs[2]);
	
	sol[1]=deti*
	  (-(mat[2][2]*mat[1][0]-mat[2][0]*mat[1][2])*rhs[0]
	   +(mat[2][2]*mat[0][0]-mat[2][0]*mat[0][2])*rhs[1]
	   -(mat[1][2]*mat[0][0]-mat[1][0]*mat[0][2])*rhs[2]);
	
	sol[2]=deti*
	  ((mat[2][1]*mat[1][0]-mat[2][0]*mat[1][1])*rhs[0]
	   -(mat[2][1]*mat[0][0]-mat[2][0]*mat[0][1])*rhs[1]
	   +(mat[1][1]*mat[0][0]-mat[1][0]*mat[0][1])*rhs[2]);
	
	if ((sol[0]+EPS1)*(sol[0]-ONE1) < 0) {
	  if ((sol[1]+EPS2) >= 0 && (sol[2]+EPS2) >=0 && 
	      (sol[1]+sol[2]) <= ONE2) 
	    {
	      xi[0]=xp[0]+sol[0]*(xx[0]-xp[0]);
	      xi[1]=xp[1]+sol[0]*(xx[1]-xp[1]);
	      xi[2]=xp[2]+sol[0]*(xx[2]-xp[2]);
	      return 1;
	    }			  
	}
      }
    }
  return 0;
}
     

      
      
	
	    
  
  

				      
				      

	


    
  
    




					
