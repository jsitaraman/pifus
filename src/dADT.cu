#include "dADT.h"
#include "pifus_types.h"
#include "pifus_cuda.h"

namespace PIFUS {

void dADT::setHostTree(int nelemInput,int *adtIntegersInput, double *adtRealsInput,
                            double *adtExtentsInput, double *coordInput)
{

nelem=nelemInput;

pushToDeviceInt(adtIntegers,adtIntegersInput,4*nelem*sizeof(int));
pushToDeviceDouble(adtReals,adtRealsInput,ndim*nelem*sizeof(double));
pushToDeviceDouble(adtExtents,adtExtentsInput,ndim*sizeof(double));
pushToDeviceDouble(coord,coordInput,ndim*nelem*sizeof(double));

/*
int isize=4*nelem*sizeof(int);
hipMalloc((void **) &adtIntegers,isize);
hipMemcpy(adtIntegers,adtIntegersInput,isize,hipMemcpyHostToDevice);

int rsize=ndim*nelem*sizeof(double);
hipMalloc((void **) &adtReals,rsize);
hipMemcpy(adtReals,adtRealsInput,rsize,hipMemcpyHostToDevice);

hipMalloc((void **) &coord,rsize);
hipMemcpy(coord,coordInput,rsize,hipMemcpyHostToDevice);

rsize=ndim*sizeof(double);
hipMalloc((void **) &adtExtents,rsize);
hipMemcpy(adtReals,adtExtentsInput,rsize,hipMemcpyHostToDevice);

rsize=ndim*sizeof(double);
hipMalloc((void **) &adtExtents,rsize);
hipMemcpy(adtExtents,adtExtentsInput,rsize,hipMemcpyHostToDevice);
*/

}


void dADT::clearData(void)
{
 if (adtIntegers) deallocateDevice(adtIntegers);
 if (adtReals) deallocateDevice(adtReals);
 if (adtExtents) deallocateDevice(adtExtents);
 if (coord) deallocateDevice(coord);
 adtIntegers=nullptr;
 adtReals=nullptr;
 adtExtents=nullptr;
}

}
