#include<hip/hip_runtime.h>
#include "dADT.h"
#include "cuda_macros.h"

void dADT::setHostTree(int nelemInput,int *adtIntegersInput, double *adtRealsInput,
                            double *adtExtentsInput, double *coordInput)
{

nelem=nelemInput;

pushToDevice(adtIntegers,adtIntegersInput,4*nelem*sizeof(int));
pushToDevice(adtReals,adtRealsInput,ndim*nelem*sizeof(double));
pushToDevice(adtExtents,adtExtentsInput,ndim*sizeof(double));
pushToDevice(coord,coordInput,ndim*nelem*sizeof(double));

/*
int isize=4*nelem*sizeof(int);
hipMalloc((void **) &adtIntegers,isize);
hipMemcpy(adtIntegers,adtIntegersInput,isize,hipMemcpyHostToDevice);

int rsize=ndim*nelem*sizeof(double);
hipMalloc((void **) &adtReals,rsize);
hipMemcpy(adtReals,adtRealsInput,rsize,hipMemcpyHostToDevice);

hipMalloc((void **) &coord,rsize);
hipMemcpy(coord,coordInput,rsize,hipMemcpyHostToDevice);

rsize=ndim*sizeof(double);
hipMalloc((void **) &adtExtents,rsize);
hipMemcpy(adtReals,adtExtentsInput,rsize,hipMemcpyHostToDevice);

rsize=ndim*sizeof(double);
hipMalloc((void **) &adtExtents,rsize);
hipMemcpy(adtExtents,adtExtentsInput,rsize,hipMemcpyHostToDevice);
*/

}


void dADT::clearData(void)
{
 if (adtIntegers) hipFree(adtIntegers);
 if (adtReals) hipFree(adtReals);
 if (adtExtents) hipFree(adtExtents);
 if (coord) hipFree(coord);
 adtIntegers=NULL;
 adtReals=NULL;
 adtExtents=NULL;
}
