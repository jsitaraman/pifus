
#include <hip/hip_runtime.h>
// -*- c++ -*- 
//#include "precision.h"
#define BIGVAL 1E6
//#include<math.h>
//#include<stdio.h>
//#include<stdlib.h>
#define REAL double
__global__ void preprocess_nb_grid(int *ndc4,
			int *ndc5,
			int *ndc6,
			int *ndc8,
			REAL *x,
			REAL *cellCenter,
			REAL *cellVol,
			REAL dsx,REAL dsy,REAL dsz,
			REAL xminx,REAL xminy,REAL xminz,
			int mdimx,int mdimy,int mdimz,
			int *itag,
			int *celltag,
			int *auxGrid,
			REAL *scfac,
			int ntetra,
			int npyra,
			int nprizm,
			int nhexa,
			int ncells)

{
  int idx=blockIdx.x*blockDim.x + threadIdx.x;
  int e[8],i,j,k,nvert,iflag;
  int icell;
  REAL xx;
  REAL axmin[3],axmax[3];
  REAL xmin[3],ds[3];
  int mdim[3];
  int ixmin[3],ixmax[3];

  if (idx < ncells)
  {
    xmin[0]=xminx;
    xmin[1]=xminy;
    xmin[2]=xminz;
    mdim[0]=mdimx;
    mdim[1]=mdimy;
    mdim[2]=mdimz;
    ds[0]=dsx;
    ds[1]=dsy;
    ds[2]=dsz;
    //    printf("%d \n",idx);
    if (idx < ntetra)
      {
	icell=4*idx;
	nvert=4;
	for(k=0;k<nvert;k++)
	  e[k]=ndc4[icell+k];
      }
    else if (idx < ntetra+npyra) 
      {
	icell=5*(idx-ntetra);
	nvert=5;
	for(k=0;k<nvert;k++)
	  e[k]=ndc5[icell+k];
      }
    else if (idx < ntetra+npyra+nprizm)
      {
	icell=6*(idx-ntetra-npyra);
	nvert=6;
	for(k=0;k<nvert;k++)
	  e[k]=ndc6[icell+k];
      }
    else if (idx < ntetra+npyra+nprizm+nhexa)
      {
	icell=8*(idx-ntetra-npyra-nprizm);
	nvert=8;
	for(k=0;k<nvert;k++)
	  e[k]=ndc8[icell+k];
      }
    
    axmin[0]=axmin[1]=axmin[2]=BIGVAL;
    axmax[0]=axmax[1]=axmax[2]=-BIGVAL;
    //
    // find the bounding box of the given cell
    //
    for(i=0;i<nvert;i++)
      for(k=0;k<3;k++)
	{
	  xx=x[3*e[i]+k];
	  axmin[k]=axmin[k] > xx ? xx : axmin[k];
	  axmax[k]=axmax[k] < xx ? xx : axmax[k];
	}
    //
    // find the index limits
    // 
    for(k=0;k<3;k++)
      {
	ixmin[k]=(int)floor(((axmin[k]-xmin[k])/ds[k]));
	ixmax[k]=(int)floor(((axmax[k]-xmin[k])/ds[k]));
      }
    //
    for(k=ixmin[2];k<=ixmax[2];k++)
      for(j=ixmin[1];j<=ixmax[1];j++)
	for (i=ixmin[0];i<=ixmax[0];i++)
	  {
	    auxGrid[k*mdim[1]*mdim[0]+j*mdim[0]+i]=idx;
	  }
    //
    // tag wall boundary cells //
    //
    iflag=0;
    for(i=0;i<nvert;i++) 
      {
	iflag=iflag + (itag[e[i]] == 1);
      }
    
    if (iflag > 0) 
      { 
	celltag[idx]=1;
      }
    else
      {
	celltag[idx]=0;
      }
    //
    // tag outer boundary nodes and cells //
    //
    iflag=0;
    for(i=0;i<nvert;i++) 
      {
	iflag=iflag + (itag[e[i]] == 2);
      }
    
    if (iflag > 0) 
      {
	scfac[idx]=BIGVAL;
      }
    else
      {
	scfac[idx]=cellVol[idx];
      }
  }
}
     

__global__ void set_tag(int *itag,int nnodes)
{
  int idx=blockIdx.x*blockDim.x + threadIdx.x;;
  if (idx < nnodes)
    {
      itag[idx]=0;
    }
}

__global__ void set_interp(int *interp,int ncells)
{
  int idx=blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < ncells)
    {
      interp[idx]=-1;
    }
}  
  
__global__ void set_wbc_tags(int *itag,int *wbcnode,int nwbc)
{
  int idx=blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < nwbc)
    {
      itag[wbcnode[idx]]=1;
    }
}			

__global__ void set_obc_tags(int *itag,int *obcnode,int nobc)
{
    int idx=blockIdx.x*blockDim.x + threadIdx.x;
    if (idx < nobc)
      {
	itag[obcnode[idx]]=2;
      }
}			

__global__ void setScalar(int N, int* vec, int val)
{
   int idx = blockIdx.x*blockDim.x + threadIdx.x;
   if (idx < N)
     {
        vec[idx]=val;
     }
}


template < typename T >
__global__ void setVector(int nr, int nc, T* vec, T val)
{
   int idx = blockIdx.x*blockDim.x + threadIdx.x;
   if (idx < nr)
    {
     vec[0+nc*idx]=val;
     vec[1+nc*idx]=val;
     vec[2+nc*idx]=val;
    }
}


