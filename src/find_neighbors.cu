
#include <hip/hip_runtime.h>
// -*- c++ -*- 
//#include "precision.h"
//#define BIGVAL 1E6
__global__ void find_neighbors(int *c2f,
	   	               int *face,
			       int *neig,
			       int ncells)
{
  int idx=blockIdx.x*blockDim.x + threadIdx.x;
  int j,faceid,c1,c2;

  if (idx < ncells) 
   {
    for(j=0;j<6;j++)
      {
	faceid=c2f[6*idx+j];
	if (faceid >=0) {
	  c1=face[2*faceid];
	  c2=face[2*faceid+1];
	  if (c1==idx) {
	    neig[6*idx+j]=c2;
	  }
	  if (c2==idx) {
	    neig[6*idx+j]=c1;
	  }
	}
      }
  }
}


